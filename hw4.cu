#include "hip/hip_runtime.h"
//***********************************************************************************
// 2018.04.01 created by Zexlus1126
//
//    Example 002
// This is a simple demonstration on calculating merkle root from merkle branch 
// and solving a block (#286819) which the information is downloaded from Block Explorer 
//***********************************************************************************

#include <iostream>
#include <fstream>
#include <string>
#include <cstdio>
#include <cstring>
#include <cassert>
#include <chrono>

#include "sha256.h"
using namespace std::chrono;
using namespace std;

#define N_TSK_EXP 32
#define N_TSK_PER_THRD_EXP 14
#define N_THRD_EXP (N_TSK_EXP - N_TSK_PER_THRD_EXP) // 18
#define N_THRD_PER_BLK_EXP 7  

// #define N_BLK (1 << (N_THRD_EXP - N_THRD_PER_BLK_EXP)) // 2048 (2560 SM's)
// #define N_THRD_PER_BLK (1 << (N_THRD_PER_BLK_EXP)) // 128
// #define N_TSK_PER_THRD (1 << (N_TSK_PER_THRD_EXP)) // 16384

// 24 sec
#define N_BLK 2048
#define N_THRD_PER_BLK 64
#define N_TSK_PER_THRD 32768

// #define N_BLK 2560
// #define N_THRD_PER_BLK 64
// #define N_TSK_PER_THRD 26215


#define SIZE_TOTAL_LSM (N_THRD_PER_BLK * 256)
#define BASE_ADDR_RAW_BLKHDR (SIZE_TOTAL_LSM >> 2)
#define BASE_ADDR_BLKHDR_COMMON_SHA ((SIZE_TOTAL_LSM + 80) >> 2)
#define BASE_ADDR_TD ((SIZE_TOTAL_LSM + 112) >> 2)
#define BASE_ADDR_k ((SIZE_TOTAL_LSM + 144) >> 2)
#define BASE_ADDR_w ((SIZE_TOTAL_LSM + 400) >> 2)

#define BASE_ADDR_THRD_LOCAL_SM 0


typedef struct
{
  BYTE b[4];
} byte_group_t;


////////////////////////   Block   /////////////////////

typedef struct _block
{
    unsigned int version;
    unsigned char prevhash[32];
    unsigned char merkle_root[32];
    unsigned int ntime;
    unsigned int nbits;
    unsigned int nonce;
}HashBlock;

#define BLK_HDR_SIZE 80

// sizeof(block) == 80 (bytes).



////////////////////////   Utils   ///////////////////////

//convert one hex-codec char to binary
unsigned char decode(unsigned char c)
{
    switch(c)
    {
        case 'a':
            return 0x0a;
        case 'b':
            return 0x0b;
        case 'c':
            return 0x0c;
        case 'd':
            return 0x0d;
        case 'e':
            return 0x0e;
        case 'f':
            return 0x0f;

        case '0' ... '9':
            return c-'0';
    }
}



// `in` is a string of 64 char's, 
    // e.g. "7938131056d5e703b8638cf3cb937755c8be0f1909f44e2a1886fbd2fbca43e0",
    // from file `casexx.in`.

// `out` is its 256-bit representation.

void convert_string_to_little_endian_bytes(unsigned char* out, char *in, size_t string_len)
{
    assert(string_len % 2 == 0);

    size_t s = 0;
    size_t b = string_len/2-1;

    for(s, b; s < string_len; s+=2, --b)
    {
        out[b] = (unsigned char)(decode(in[s])<<4) + decode(in[s+1]);
    }
}



// // print out binary array (from highest value) in the hex format
// void print_hex(unsigned char* hex, size_t len)
// {
//     for(int i=0;i<len;++i)
//     {
//         printf("%02x", hex[i]);
//     }
// }



__device__ void print_hex(unsigned char* hex, size_t n_bytes)
{
    printf("0x");
    for(int i=n_bytes-1;i>=0;i--)
    {
        printf("%02x", hex[i]);
    }
    
}

// print out binar array (from lowest value) in the hex format
void print_hex_inverse(unsigned char* hex, size_t len)
{
    for(int i=len-1;i>=0;--i)
    {
        printf("%02x", hex[i]);
    }
}


void getline(char *str, size_t len, FILE *fp)
{

    int i=0;
    while( i<len && (str[i] = fgetc(fp)) != EOF && str[i++] != '\n');
    str[len-1] = '\0';
}


void double_sha256(SHA256 *sha256_ctx, unsigned char *bytes, size_t len)
{
    SHA256 tmp;

    // tmp = hash(list[i]+list[i+1]), i: 0 ~ txLen - 1, i += 2.
    sha256(&tmp, (BYTE*)bytes, len);   
    
    // list[j] = hash(tmp), j: 0 ~ txLen / 2 - 1, j += 1.    
    sha256(sha256_ctx, (BYTE*)&tmp, sizeof(tmp));
}

void calc_merkle_root(unsigned char *root, int count, char **branch)
{
    size_t total_count = count; // merkle branch
    unsigned char *raw_list = new unsigned char[(total_count+1)*32]; // `*32`: 256 bits == 32 bytes.
    unsigned char **list = new unsigned char*[total_count+1];

    // copy each branch to the list
    for(int i=0; i < total_count; ++i)
    {
        list[i] = raw_list + i * 32;
        convert_string_to_little_endian_bytes(list[i], branch[i], 64);
    }

    list[total_count] = raw_list + total_count * 32;


    // calculate merkle root
    while(total_count > 1)
    {
        
        // hash each pair
        int i, j;

        if(total_count % 2 == 1)  //odd, 
        {
            // void *memcpy(void *dest, const void * src, size_t n).
            memcpy(list[total_count], list[total_count-1], 32);
        }



        for(i=0, j=0; i < total_count; i += 2, ++j)
        {

            // double_sha:
            //     tmp = hash(list[0]+list[1])
            //     list[0] = hash(tmp)
            double_sha256((SHA256*)list[j], list[i], 64);
        }

        total_count = j; // halved in each iteration.
    }

    memcpy(root, list[0], 32);

    delete[] raw_list;
    delete[] list;
}






// ###############################################################################


// __device__ 
// int little_endian_bit_comparison_dev(const unsigned char *a, 
//                                         const unsigned char *b){
    
__device__ 
int little_endian_bit_comparison_dev(const byte_group_t *a, 
                                        const unsigned char *b){
    for(int i = 7; i >= 0; --i)
    {
        for(int j = 0; j <= 3; ++j){
            
            if(a[N_THRD_PER_BLK * i].b[j] < b[4 * i + 3 - j])
                return -1;
            else if(a[N_THRD_PER_BLK * i].b[j] > b[4 * i + 3 - j])
                return 1;
        }
    }
    
    return 0;
}


__device__
void sha256_transform_dev(SHA256 *ctx, const BYTE *msg){
	
    WORD a, b, c, d, e, f, g, h;
	WORD i, j;
	

	WORD w[64];

	for(i=0, j=0; i < 16; ++i, j += 4)
	{
		w[i] = (msg[j]<<24) | (msg[j+1]<<16) | (msg[j+2]<<8) | (msg[j+3]);
	}
	

	for( i = 16; i < 64; ++i)
	{
		WORD s0 = (_rotr(w[i-15], 7)) ^ (_rotr(w[i-15], 18)) ^ (w[i-15] >> 3);
		WORD s1 = (_rotr(w[i-2], 17)) ^ (_rotr(w[i-2], 19))  ^ (w[i-2] >> 10);
		w[i] = w[i-16] + s0 + w[i-7] + s1;
	}
	

	a = ctx->h[0];
	b = ctx->h[1];
	c = ctx->h[2];
	d = ctx->h[3];
	e = ctx->h[4];
	f = ctx->h[5];
	g = ctx->h[6];
	h = ctx->h[7];
	

	for(i=0;i<64;++i)
	{
		WORD S0 = (_rotr(a, 2)) ^ (_rotr(a, 13)) ^ (_rotr(a, 22));
		WORD S1 = (_rotr(e, 6)) ^ (_rotr(e, 11)) ^ (_rotr(e, 25));
		WORD ch = (e & f) ^ ((~e) & g);
		WORD maj = (a & b) ^ (a & c) ^ (b & c);
		WORD temp1 = h + S1 + ch + k_dev[i] + w[i];
		WORD temp2 = S0 + maj;
		
		h = g;
		g = f;
		f = e;
		e = d + temp1;
		d = c;
		c = b;
		b = a;
		a = temp1 + temp2;
	}


	ctx->h[0] += a;
	ctx->h[1] += b;
	ctx->h[2] += c;
	ctx->h[3] += d;
	ctx->h[4] += e;
	ctx->h[5] += f;
	ctx->h[6] += g;
	ctx->h[7] += h;
	
}











__device__ 
void sha256_commonBlkhdr_dev(SHA256 *ctx, const BYTE *msg){

	ctx->h[0] = 0x6a09e667;
	ctx->h[1] = 0xbb67ae85;
	ctx->h[2] = 0x3c6ef372;
	ctx->h[3] = 0xa54ff53a;
	ctx->h[4] = 0x510e527f;
	ctx->h[5] = 0x9b05688c;
	ctx->h[6] = 0x1f83d9ab;
	ctx->h[7] = 0x5be0cd19;

	sha256_transform_dev(ctx, &msg[0]);
}





__device__ 
void compute_w(byte_group_t *sm){
    
    WORD i;

    byte_group_t *msg = &sm[BASE_ADDR_THRD_LOCAL_SM + threadIdx.x];


    #pragma unroll
    for(int i=0; i < 16; i++){
        ((WORD *)&msg[N_THRD_PER_BLK * i])[0] = 0;
    }

    WORD *w = (WORD *)&sm[BASE_ADDR_w];


    // ((WORD *)(&msg[N_THRD_PER_BLK * 3]))[0] = nonce;


    #pragma unroll
	for(i=0; i < 3; ++i) 
	{
        ((WORD *)&msg[N_THRD_PER_BLK * i])[0] = \
                    ((WORD *)&sm[BASE_ADDR_RAW_BLKHDR])[i + 16];
	}

  
    msg[N_THRD_PER_BLK * 4].b[0] = 0x80;  

    msg[N_THRD_PER_BLK * 15].b[3] = 640;  
    msg[N_THRD_PER_BLK * 15].b[2] = 2;  



    #pragma unroll
	for(i=0; i < 5; ++i)
	{
		w[i] = (msg[N_THRD_PER_BLK * i].b[0]<<24) | \
                                (msg[N_THRD_PER_BLK * i].b[1]<<16)| \
                                (msg[N_THRD_PER_BLK * i].b[2]<<8) | \
                                (msg[N_THRD_PER_BLK * i].b[3]);
	}




    #pragma unroll
	for(; i < 15; ++i)
	{
		w[i] = 0;
	}

    w[i] = (msg[N_THRD_PER_BLK * i].b[0]<<24) | \
                            (msg[N_THRD_PER_BLK * i].b[1]<<16)| \
                            (msg[N_THRD_PER_BLK * i].b[2]<<8) | \
                            (msg[N_THRD_PER_BLK * i].b[3]);
}



__device__ 
void sha256_stage1_dev(byte_group_t *sm, unsigned int nonce){

	WORD i;
    WORD a, b, c, d, e, f, g, h, t1, t2;

    WORD *k = (WORD *)&sm[BASE_ADDR_k];

    WORD *w = (WORD *)&sm[BASE_ADDR_THRD_LOCAL_SM + threadIdx.x];

    #pragma unroll
    for(i=0; i<16; i++) w[N_THRD_PER_BLK * i] = ((WORD *)&sm[BASE_ADDR_w])[i];
    w[N_THRD_PER_BLK * 3] = nonce;

    // printf("%d -> %d\n", nonce, (_rotr(nonce, 7)) ^ (_rotr(nonce, 18)) ^ (nonce >> 3));



    int gtid = blockIdx.x * blockDim.x + threadIdx.x;


    // ~ 2300 cycles
    #pragma unroll
	for( i = 16; i < 64; ++i)
	{

        // 16 <- 0, 1, 9, 14 
        // 17 <- 1, 2, 10, 15
        // 18* <- 2, 3*, 11, 16
        // 19* <- 3*, 4, 12, 17 
        // 20* <- 4, 5, 13, 18*
        // 21* <- 5, 6, 14, 19*
        // 22* <- 6, 7, 15, 20*
        // 23* <- 7, 8, 16, 21*
        // 24* <- 8, 9, 17, 22*
        // 25* <- 9, 10, 18*, 23*

        c = w[N_THRD_PER_BLK * (i-16)];
        a = w[N_THRD_PER_BLK * (i-15)];
        d = w[N_THRD_PER_BLK * (i-7)];
        b = w[N_THRD_PER_BLK * (i-2)];
        
		WORD s0 = (_rotr(a, 7)) ^ (_rotr(a, 18)) ^ (a >> 3);
		WORD s1 = (_rotr(b, 17)) ^ (_rotr(b, 19)) ^ (b >> 10);

		w[N_THRD_PER_BLK * i] = c + s0 + d + s1;     
	}




	a = ((WORD *)&sm[BASE_ADDR_BLKHDR_COMMON_SHA])[0];
	b = ((WORD *)&sm[BASE_ADDR_BLKHDR_COMMON_SHA])[1];
	c = ((WORD *)&sm[BASE_ADDR_BLKHDR_COMMON_SHA])[2];
	d = ((WORD *)&sm[BASE_ADDR_BLKHDR_COMMON_SHA])[3];
	e = ((WORD *)&sm[BASE_ADDR_BLKHDR_COMMON_SHA])[4];
	f = ((WORD *)&sm[BASE_ADDR_BLKHDR_COMMON_SHA])[5];
	g = ((WORD *)&sm[BASE_ADDR_BLKHDR_COMMON_SHA])[6];
	h = ((WORD *)&sm[BASE_ADDR_BLKHDR_COMMON_SHA])[7];

    SHA256_COMPRESS_8X

	w[N_THRD_PER_BLK * 0] = ((WORD *)&sm[BASE_ADDR_BLKHDR_COMMON_SHA])[0] + a;
	w[N_THRD_PER_BLK * 1] = ((WORD *)&sm[BASE_ADDR_BLKHDR_COMMON_SHA])[1] + b;
	w[N_THRD_PER_BLK * 2] = ((WORD *)&sm[BASE_ADDR_BLKHDR_COMMON_SHA])[2] + c;
	w[N_THRD_PER_BLK * 3] = ((WORD *)&sm[BASE_ADDR_BLKHDR_COMMON_SHA])[3] + d;
	w[N_THRD_PER_BLK * 4] = ((WORD *)&sm[BASE_ADDR_BLKHDR_COMMON_SHA])[4] + e;
	w[N_THRD_PER_BLK * 5] = ((WORD *)&sm[BASE_ADDR_BLKHDR_COMMON_SHA])[5] + f;
	w[N_THRD_PER_BLK * 6] = ((WORD *)&sm[BASE_ADDR_BLKHDR_COMMON_SHA])[6] + g;
	w[N_THRD_PER_BLK * 7] = ((WORD *)&sm[BASE_ADDR_BLKHDR_COMMON_SHA])[7] + h;
    

}









__device__ 
void sha256_stage2_dev(byte_group_t *sm){
	
	WORD i;
    WORD a, b, c, d, e, f, g, h, t1, t2;	

    WORD *k = (WORD *)&sm[BASE_ADDR_k];

    WORD *w = (WORD *)&sm[BASE_ADDR_THRD_LOCAL_SM + threadIdx.x];
    
    w[N_THRD_PER_BLK * 8] = 0x80000000;  
    w[N_THRD_PER_BLK * 15] = 0x00000100;  

    #pragma unroll
	for(i = 9; i < 15; ++i) w[N_THRD_PER_BLK * i] = 0;

    #pragma unroll
	for( i = 16; i < 64; ++i)
	{
        c = w[N_THRD_PER_BLK * (i-16)];
        a = w[N_THRD_PER_BLK * (i-15)];
        d = w[N_THRD_PER_BLK * (i-7)];
        b = w[N_THRD_PER_BLK * (i-2)];

		WORD s0 = (_rotr(a, 7)) ^ (_rotr(a, 18)) ^  (a >> 3);
		WORD s1 = (_rotr(b, 17)) ^ (_rotr(b, 19)) ^ (b >> 10);

		w[N_THRD_PER_BLK * i] = c + s0 + d + s1;
	}


	a = 0x6a09e667;
	b = 0xbb67ae85;
	c = 0x3c6ef372;
	d = 0xa54ff53a;
	e = 0x510e527f;
	f = 0x9b05688c;
	g = 0x1f83d9ab;
	h = 0x5be0cd19;
    

    SHA256_COMPRESS_8X


	w[N_THRD_PER_BLK * 0] = a + 0x6a09e667;
	w[N_THRD_PER_BLK * 1] = b + 0xbb67ae85;
	w[N_THRD_PER_BLK * 2] = c + 0x3c6ef372;
	w[N_THRD_PER_BLK * 3] = d + 0xa54ff53a;
	w[N_THRD_PER_BLK * 4] = e + 0x510e527f;
	w[N_THRD_PER_BLK * 5] = f + 0x9b05688c;
	w[N_THRD_PER_BLK * 6] = g + 0x1f83d9ab;
	w[N_THRD_PER_BLK * 7] = h + 0x5be0cd19;
}




__device__ 
void compute_target_difficulty(byte_group_t *sm){
    
    HashBlock *blk = (HashBlock *)&sm[BASE_ADDR_RAW_BLKHDR];

    unsigned int exp = blk->nbits >> 24;
    unsigned int mant = blk->nbits & 0xffffff;
    
    unsigned int shift = 8 * (exp - 3);
    unsigned int sb = shift >> 3; 
    unsigned int rb = shift % 8; 
    
    for(int i=0;i<8;i++){
        ((WORD *)&sm[BASE_ADDR_TD])[i] = 0;
    }

    ((BYTE *)&sm[BASE_ADDR_TD])[sb    ] = (mant << rb);      
    ((BYTE *)&sm[BASE_ADDR_TD])[sb + 1] = (mant >> (8-rb));  
    ((BYTE *)&sm[BASE_ADDR_TD])[sb + 2] = (mant >> (16-rb)); 
    ((BYTE *)&sm[BASE_ADDR_TD])[sb + 3] = (mant >> (24-rb));    

}




__global__ void nonceSearch(unsigned char *blockHeader, unsigned int *nonceValidDev, int d, int n)
{

    int gtid = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    int nlz_target;

    __shared__ byte_group_t sm[(SIZE_TOTAL_LSM + 80 + 32 + 32 + 256 + 256) / 4];

    
    // clock_t start_time = clock(); 
    // clock_t stop_time = clock();
    // int runtime = (int)(stop_time - start_time);
    // if(gtid == 0){
    //     printf("load sm dt: %d\n", runtime);
    // }


    if(tid < 20){
        ((WORD *)&sm[BASE_ADDR_RAW_BLKHDR])[tid] = ((WORD *)blockHeader)[tid];
    }

    if(tid < 64){
        ((WORD *)&sm[BASE_ADDR_k])[tid] = ((WORD *)k_dev)[tid];
    }


    __syncthreads();


    if(tid == 0){
        sha256_commonBlkhdr_dev((SHA256 *)&sm[BASE_ADDR_BLKHDR_COMMON_SHA], 
                                                (BYTE *)&sm[BASE_ADDR_RAW_BLKHDR]);
        compute_target_difficulty(sm);
        compute_w(sm);
    }

    __syncthreads();


    unsigned int nonce;
    

    for(nonce = gtid * N_TSK_PER_THRD + (N_TSK_PER_THRD / d) * n; 
        nonce < gtid * N_TSK_PER_THRD + (N_TSK_PER_THRD / d) * (n + 1); 
        ++nonce) 
    {       
        sha256_stage1_dev(sm, nonce);
        sha256_stage2_dev(sm); 

        if(little_endian_bit_comparison_dev(&sm[BASE_ADDR_THRD_LOCAL_SM + threadIdx.x], 
                                (BYTE *)&sm[BASE_ADDR_TD]) < 0)  
        {

            *nonceValidDev = (((BYTE *)&nonce)[0]<<24) | \
                                    (((BYTE *)&nonce)[1]<<16)| \
                                    (((BYTE *)&nonce)[2]<<8) | \
                                    (((BYTE *)&nonce)[3]);

            break;
        }     
    }
}




void solve(FILE *fin, FILE *fout)
{

    // **** read data *****
    char version[9];
    char prevhash[65];
    char ntime[9];
    char nbits[9];
    int tx;
    char *raw_merkle_branch;
    char **merkle_branch;


    auto start = high_resolution_clock::now();

    getline(version, 9, fin);   
    getline(prevhash, 65, fin); 
    getline(ntime, 9, fin);
    getline(nbits, 9, fin);
    fscanf(fin, "%d\n", &tx);

    raw_merkle_branch = new char [tx * 65];
    merkle_branch = new char *[tx];

    for(int i = 0; i < tx; ++i)
    {
        merkle_branch[i] = raw_merkle_branch + i * 65;
        getline(merkle_branch[i], 65, fin);
        merkle_branch[i][64] = '\0';
    }

    unsigned char merkle_root[32];
    
    calc_merkle_root(merkle_root, tx, merkle_branch);

    HashBlock block;
  
    convert_string_to_little_endian_bytes((unsigned char *)&block.version, version, 8);
    convert_string_to_little_endian_bytes(block.prevhash,                  prevhash,    64);
    memcpy(block.merkle_root, merkle_root, 32);
    convert_string_to_little_endian_bytes((unsigned char *)&block.nbits,   nbits,     8);
    convert_string_to_little_endian_bytes((unsigned char *)&block.ntime,   ntime,     8);
    block.nonce = 0;

    unsigned char *blockHeaderDev;
    unsigned int *nonceValidDev;
    unsigned int nonceValidHost = 0;

    hipMalloc(&blockHeaderDev, BLK_HDR_SIZE);
    hipMemcpy(blockHeaderDev, (unsigned char*)&block,
                         BLK_HDR_SIZE, hipMemcpyHostToDevice);

    hipMalloc(&nonceValidDev, sizeof(int));
    hipMemset(nonceValidDev, 0, sizeof(int));
    
    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);
    cout<<"read tx & calc_merkle_root() time: "<<duration.count()/ 1000000.0 <<" sec"<<endl;


    int d = 16;

    for(int i=0;i<d;i++){

        nonceSearch<<< N_BLK, N_THRD_PER_BLK >>> (blockHeaderDev, nonceValidDev, d, i); 
        hipDeviceSynchronize();
        hipMemcpy(&nonceValidHost, nonceValidDev, sizeof(int), hipMemcpyDeviceToHost);
        
        if(nonceValidHost) break;     
    }
    

    for(int i=0; i < 4; ++i)
    {
        fprintf(fout, "%02x", ((unsigned char *)&nonceValidHost)[i]);
    }
    fprintf(fout, "\n");

    

    delete[] merkle_branch;
    delete[] raw_merkle_branch;
}







int main(int argc, char **argv)
{
    // hipDeviceSetCacheConfig(hipFuncCachePreferL1);

    

    if (argc != 3) {
        fprintf(stderr, "usage: cuda_miner <in> <out>\n");
    }

    FILE *fin = fopen(argv[1], "r");
    FILE *fout = fopen(argv[2], "w");

    int totalblock;

    fscanf(fin, "%d\n", &totalblock);
    fprintf(fout, "%d\n", totalblock);

    for(int i=0; i < totalblock; ++i)
    {
        solve(fin, fout);
    }

    return 0;
}